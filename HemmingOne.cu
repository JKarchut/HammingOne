
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

void parseNumber(unsigned int *arr, std::string number)
{
    int bitNum = sizeof(unsigned int) * 8;
    int bitPos = 0;
    int arrPos = 0;
    unsigned int pomValue = 0;
    for(int x = 0; x < number.length();x++)
    {
        if(bitNum > bitPos)
        {
            pomValue+= (number[x] - '0')<<bitPos;
            bitPos++;
        }
        else
        {
            arr[arrPos] = pomValue;
            arrPos++;
            bitPos = 0;
        }
    }
}

int main(int argc, char** argv)
{
    std::ifstream data(argv[1]);
    int n;
    int l;
    data >> n;
    data >> l;
    unsigned int** arr = new unsigned int*[n];
    for(int x = 0 ; x < n; x++)
    {
        arr[x] = new unsigned int[l];
    }
    std::string number;
    int arrPos = 0;
    while(data >> number)
    {
        parseNumber(arr[arrPos], number);
        arrPos++;
    }
    int taken = l / (sizeof(unsigned int) * 8);
    for(int x =0 ; x<n; x++)
    {
        for (int y = 0; y < taken; y++)
        {
            std::cout << arr[x][y] << ' ';
        }
        std::cout << std::endl;
    }
    data.close();
    for(int x = 0; x < n; x++)
    {
        delete[] arr[x];
    }
    delete[] arr;
}