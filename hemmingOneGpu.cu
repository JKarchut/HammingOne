
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstring>
void parseNumber(unsigned int *arr, std::string number, int bitsPerInt)
{
    int bitPos = 0;
    int arrPos = 0;
    unsigned int pomValue = 0;
    for(int x = 0; x < number.length();x++)
    {
        if(bitsPerInt > bitPos)
        {
            pomValue += ((unsigned int)(number[x] - '0') << bitPos);
            bitPos++;
        }
        else
        {
            arr[arrPos] = pomValue;
            arrPos++;
            bitPos = 0;
            pomValue = 0;
        }
    }
    if(bitPos != 0)
    {
        arr[arrPos] = pomValue;
    }
}

__global__ void findPairs(unsigned int *arr, int n, int l)
{
    int id = blockIdx.x * 1024 + threadIdx.x;
    if (id >= n) return;
    int diff, pom;
    for(int x = id + 1; x < n; x++)
    {
        diff = 0;
        for(int y = 0; y < l; y++)
        {
            pom = (arr[id * l + y]^arr[x * l + y]);    
            if(pom > 0 && (pom & (pom - 1)) == 0)
                diff++;
            else if(pom > 0)
                diff = 2;

            if(diff > 1)
                break;
        }
        if(diff == 1)
        {
            printf("%d %d\n",id,x);
        }
    }
    
}

int32_t main(int argc, char** argv)
{
    std::ifstream data(argv[1]);
    int n;
    int l;
    data >> n;
    data >> l;
    int bitsPerInt =  15;
    int taken = l / bitsPerInt;
    if(l % bitsPerInt != 0)
        taken++;
    unsigned int* arr = new unsigned int[n * taken];
    memset(arr,0,taken * n * sizeof(unsigned int));
    std::string number;
    int arrPos = 0;
    while(data >> number)
    {
        parseNumber(&arr[taken * arrPos], number, bitsPerInt);
        arrPos++;
    }

    unsigned int* arr_d;
    hipMalloc(&arr_d, n * taken * sizeof(unsigned int));
    hipMemcpy(arr_d,arr, n * taken * sizeof(unsigned int), hipMemcpyHostToDevice);
    int threadCount = 1024;
    int blockSize = n / threadCount + 1;
    findPairs<<<blockSize,threadCount>>>(arr_d,n,taken);
    data.close();
    hipFree(arr_d);
    delete[] arr;
}