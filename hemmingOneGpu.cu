#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstring>
 
void parseNumber(  int *arr, std::string number, int bitsPerInt)
{
    int bitPos = 0;
    int arrPos = 0;
      int pomValue = 0;
    for(int x = 0; x < number.length();x++)
    {
        if(bitPos < bitsPerInt)
        {
            pomValue = pomValue * 2 + ((  int)(number[x] - '0'));
            bitPos++;
        }
        else
        {
            arr[arrPos] = pomValue;
            arrPos++;
            bitPos = 0;
            pomValue = 0;
        }
    }
    if(bitPos != 0)
    {
        arr[arrPos] = pomValue;
    }
}

__global__ void findPairs(  int *arr, int n, int l)
{
    int id = blockIdx.x * 1024 + threadIdx.x;
    if (id >= n) return;
    int diff, pom;
    for(int x = id + 1; x < n; x++)
    {
        diff = 0;
        for(int y = 0; y < l; y++)
        {
            pom = (arr[id * l + y]^arr[x * l + y]);    
            while(pom != 0)
            {
                diff += (pom & 1);
                pom = (pom >> 1);
            }

            if(diff > 1)
                break;
        }
        if(diff == 1)
        {
            printf("%d %d\n",id,x);
        }
    }
    
}
int_ceil(double variable) {

int new_variable = (int)variable;

if ((double)new_variable == variable) return new_variable;

else return new_variable + 1;

}

int32_t main(int argc, char** argv)
{
    std::ifstream data(argv[1]);
    int n;
    int l;
    data >> n;
    data >> l;
    int bitsPerInt =  15;
    // int taken = l / bitsPerInt;
    // if(l % bitsPerInt != 0)
    //     taken++;
int taken = _ceil((double)l / bitsPerInt);
      int* arr = new   int[n * taken];
    memset(arr,0,taken * n * sizeof(  int));
    std::string number;
    int arrPos = 0;
    while(data >> number)
    {
        parseNumber(&arr[taken * arrPos], number, bitsPerInt);
        arrPos++;
    }

    int* arr_d;
    hipMalloc(&arr_d, n * taken * sizeof(  int));
    hipMemcpy(arr_d,arr, n * taken * sizeof(  int), hipMemcpyHostToDevice);
    int threadCount = 1024;
    // int blockSize = n / threadCount + 1;
    int blocks = _ceil((double)n / threadCount);
    findPairs<<<blocks,threadCount>>>(arr_d,n,taken);
    data.close();
    hipFree(arr_d);
    delete[] arr;
}