
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <sys/time.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void parseNumber(int *arr, std::string number, int bitsPerInt)
{
    int bitPos = 0;
    int arrPos = 0;
    int pomValue = 0;
    for(int x = 0; x < number.length();x++)
    {
        if(bitsPerInt > bitPos)
        {
            pomValue += (int)(number[x] - '0') << bitPos;
            bitPos++;
        }
        else
        {
            arr[arrPos] = pomValue;
            arrPos++;
            bitPos = 0;
            pomValue = 0;
            x--;
        }
    }
    if(bitPos != 0)
    {
        arr[arrPos] = pomValue;
    }
}

__global__ void findPairs(int *arr, int n, int l)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int diff, pom;
    for(int x = index + 1; x < n; x++)
    {
        diff = 0;
        for(int y = 0; y < l; y++)
        {
            pom = (arr[index * l + y]^arr[x * l + y]);
            if(pom != 0 && (pom & (pom - 1)) == 0)
            {
                diff++;
            }
            else if(pom > 0)
            {
                diff = 2;
            }
            if(diff > 1)
            {
                break;
            }
        }
        if(diff == 1)
        {
            printf("%d %d\n",index,x);
        }
    }
}

int32_t main(int argc, char** argv)
{
    std::ifstream data(argv[1]);
    std::ofstream measures(argv[2], std::ios::app);
    struct timeval begin, end;
    int n;
    int l;
    data >> n;
    data >> l;
    int bitsPerInt =  (sizeof(int) * 8) - 1;
    int taken = l / bitsPerInt;
    if(l % bitsPerInt != 0)
        taken++;
    int *arr;
    hipMallocManaged(&arr, (long)n * sizeof(int) * taken);
    memset(arr,0,(long)taken * n * sizeof(int));
    std::string number;
    int arrPos = 0;
    while(data >> number)
    {
        parseNumber(&arr[(long)arrPos * taken], number, bitsPerInt);
        arrPos++;
    }
    int threadCount = 1024;
    int blockCount = (n / 1024) + 1;
    gpuErrchk(hipDeviceSetLimit(hipLimitPrintfFifoSize, (long long)1e15));
    gettimeofday(&begin, 0);
    findPairs<<<blockCount,threadCount>>>(arr,n,taken);
    gettimeofday(&end, 0);
    long seconds = end.tv_sec - begin.tv_sec;
    long microseconds = end.tv_usec - begin.tv_usec;
    double elapsed = seconds + microseconds*1e-6;
    measures <<"GPU " << elapsed << "s " << std::endl;
    gpuErrchk( hipPeekAtLastError());
    gpuErrchk( hipDeviceSynchronize());
    data.close();
    measures.close();
    hipFree(arr);
    return 0;
}