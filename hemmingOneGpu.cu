#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

void parseNumber(unsigned int *arr, std::string number, int bitNum)
{
    int bitPos = 0;
    int arrPos = 0;
    unsigned int pomValue = 0;
    for(int x = 0; x < number.length();x++)
    {
        if(bitNum > bitPos)
        {
            pomValue += (unsigned int)(number[x] - '0') << bitPos;
            bitPos++;
        }
        else
        {
            arr[arrPos] = pomValue;
            arrPos++;
            bitPos = 0;
            pomValue = 0;
        }
    }
    if(bitPos != 0)
    {
        arr[arrPos] = pomValue;
    }
}

__global__ void findPairs(unsigned int *arr, int n, int l)
{
    int id = blockIdx.x;
    int comp1 = id / n;
    int comp2 = id - comp1 * n;
    int diff = 0;
    for(int x = 0; x < l; x++)
    {
        diff += arr[comp1 * n + x^arr[comp2 * n +x];    
        if(diff > 1)
        {
            break;
        }
    }
    if(diff <= 1)
    {
        printf("%d\n%d\n",comp1,comp2);
    }
}

int main(int argc, char** argv)
{
    std::ifstream data(argv[1]);
    int n;
    int l;
    data >> n;
    data >> l;
    int bitsPerInt =  sizeof(unsigned int) * 8;
    int taken = l / bitsPerInt;
    if(l % bitsPerInt != 0)
        taken++;
    unsigned int* arr = new unsigned int[n * taken];
    std::string number;
    int arrPos = 0;
    while(data >> number)
    {
        parseNumber(&arr[taken * arrPos], number, bitsPerInt);
        arrPos++;
    }

    unsigned int* arr_d;
    hipMalloc(&arr_d, n * taken * sizeof(unsigned int));
    hipMemcpy(arr_d,arr, n * taken, hipMemcpyHostToDevice);
    dim3 blockSize(n*n,1,1);
    findPairs<<<blockSize,1>>>(arr_d,n,taken);
    data.close();
    hipFree(arr_d);
    delete[] arr;
}