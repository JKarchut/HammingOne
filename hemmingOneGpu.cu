
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstring>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void parseNumber(unsigned int *arr, std::string number, int bitsPerInt)
{
    int bitPos = 0;
    int arrPos = 0;
    unsigned int pomValue = 0;
    for(int x = 0; x < number.length();x++)
    {
        if(bitsPerInt > bitPos)
        {
            pomValue += (unsigned int)(number[x] - '0') << bitPos;
            bitPos++;
        }
        else
        {
            arr[arrPos] = pomValue;
            arrPos++;
            bitPos = 0;
            pomValue = 0;
        }
    }
    if(bitPos != 0)
    {
        arr[arrPos] = pomValue;
    }
}

__global__ void findPairs(unsigned int *arr, unsigned int* ans, int n, int l)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int diff, pom;
    for(int x = index + 1; x < n; x++)
    {
        diff = 0;
        for(int y = 0; y < l; y++)
        {
            pom = (arr[index * l + y]^arr[x * l + y]);
            if(pom > 0 && (pom & (pom - 1)) == 0)
                diff++;
            else if(pom > 0)
                diff = 2;
            if(diff > 1)
                break;
        }
        if(diff == 1)
        {
            ans[index * n + x] = 1;
        }
    }
}

int32_t main(int argc, char** argv)
{
    std::ifstream data(argv[1]);
    int n;
    int l;
    data >> n;
    data >> l;
    int bitsPerInt =  sizeof(unsigned int) * 8 - 1;
    int taken = l / bitsPerInt;
    if(l % bitsPerInt != 0)
        taken++;
    unsigned int *arr, *ans;
    hipMallocManaged(&arr, (long)n * sizeof(unsigned int) * taken);
    hipMallocManaged(&ans, (long)n * sizeof(unsigned int) * n);
    memset(arr,0,(long)taken * n * sizeof(unsigned int));
    std::string number;
    int arrPos = 0;
    while(data >> number)
    {
        parseNumber(&arr[(long)arrPos * taken], number, bitsPerInt);
        arrPos++;
    }
    int threadCount = 1024;
    int blockCount = (n / 1024) + 1;
    gpuErrchk(hipDeviceSetLimit(hipLimitPrintfFifoSize, sizeof(unsigned int) * n * (n + 1)));
    findPairs<<<blockCount,threadCount>>>(arr, ans,n,taken);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    for(int x = 0; x< n;x++)
    {
        for(int y = 0; y < n; y++)
        {
            if(ans[x * n + y] == 1)
            {
                std::cout<<x<<' '<<y<<std::endl;
            }
        }
    }
    data.close();
    hipFree(arr);
    return 0;
}