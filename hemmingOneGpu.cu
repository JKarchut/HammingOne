#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstring>
void parseNumber(unsigned int *arr, std::string number, int bitsPerInt)
{
    int bitPos = 0;
    int arrPos = 0;
    unsigned int pomValue = 0;
    for(int x = 0; x < number.length();x++)
    {
        if(bitsPerInt > bitPos)
        {
            pomValue += (unsigned int)(number[x] - '0') << bitPos;
            bitPos++;
        }
        else
        {
            arr[arrPos] = pomValue;
            arrPos++;
            bitPos = 0;
            pomValue = 0;
        }
    }
    if(bitPos != 0)
    {
        arr[arrPos] = pomValue;
    }
}

__global__ void findPairs(unsigned int *arr, int n, int l)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int diff, pom;
    for(int x = index + 1; x < n; x++)
    {
        diff = 0;
        for(int y = 0; y < l; y++)
        {
            pom = (arr[index * l + y]^arr[x * l + y]);
            if(pom > 0 && (pom & (pom - 1)) == 0)
                diff++;
            else if(pom > 0)
                diff = 2;
            if(diff > 1)
                break;
        }
        if(diff == 1)
        {
            printf("%d %d\n",id,x);
        }
    }
}

int32_t main(int argc, char** argv)
{
    std::ifstream data(argv[1]);
    int n;
    int l;
    data >> n;
    data >> l;
    int bitsPerInt =  sizeof(unsigned int) * 8 - 1;
    int taken = l / bitsPerInt;
    if(l % bitsPerInt != 0)
        taken++;
    unsigned int* arr;
    hipMallocManaged(&arr, n * sizeof(unsigned int) * taken);
    memset(arr,0,taken * n * sizeof(unsigned int));
    std::string number;
    int arrPos = 0;
    while(data >> number)
    {
        parseNumber(&arr[arrPos * taken], number, bitsPerInt);
        arrPos++;
    }
    int threadCount = 1024;
    int blockCount = (n / 1024) + 1;
    hipDeviceSetLimit(hipLimitPrintfFifoSize, sizeof(unsigned int) * n * n);
    findPairs<<<blockCount,threadCount>>>(arr,n,taken);
    hipDeviceSynchronize();
    data.close();
    hipFree(arr);
    return 0;
}