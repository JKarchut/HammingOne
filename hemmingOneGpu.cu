
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

void parseNumber(unsigned int *arr, std::string number)
{
    int bitNum = sizeof(unsigned int) * 8;
    int bitPos = 0;
    int arrPos = 0;
    unsigned int pomValue = 0;
    for(int x = 0; x < number.length();x++)
    {
        if(bitNum > bitPos)
        {
            pomValue += (unsigned int)(number[x] - '0') << bitPos;
            bitPos++;
        }
        else
        {
            arr[arrPos] = pomValue;
            arrPos++;
            bitPos = 0;
            pomValue = 0;
        }
    }
    if(bitPos != 0)
    {
        arr[arrPos] = pomValue;
    }
}

__global__ void findPairs(unsigned int **arr, int n, int l)
{
    int id = blockIdx.x;
    int comp1 = id / n;
    int comp2 = id - comp1 * n;
    int diff = 0;
    for(int x = 0; x < l; x++)
    {
        diff += arr[comp1][x]^arr[comp2][x];
        if(diff > 1)
        {
            break;
        }
    }
    if(diff <= 1)
    {
        printf("%d\n%d\n",comp1,comp2);
    }
}

int main(int argc, char** argv)
{
    std::ifstream data(argv[1]);
    int n;
    int l;
    data >> n;
    data >> l;
    int taken = l / (sizeof(unsigned int) * 8);
    if(l % (sizeof(unsigned int) * 8) != 0)
        taken++;
    unsigned int** arr;
    hipMallocManaged(&arr, n * sizeof(unsigned int*));
    for(int x = 0 ; x < n; x++)
    {
        hipMallocManaged(&arr[x],taken);
    }
    std::string number;
    int arrPos = 0;
    while(data >> number)
    {
        parseNumber(arr[arrPos], number);
        arrPos++;
    }
    dim3 blockSize(n*n,1,1);
    findPairs<<<blockSize,1>>>(arr,n,taken);
    data.close();
    for(int x = 0; x < n; x++)
    {
        delete[] arr[x];
    }
    delete[] arr;
}