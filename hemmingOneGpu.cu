
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstring>
 
void parseNumber(  int *arr, std::string number, int bitsPerInt)
{
    int bitPos = 0;
    int arrPos = 0;
    int pomValue = 0;
    for(int x = 0; x < number.length();x++)
    {
        if(bitPos < bitsPerInt)
        {
            pomValue = (pomValue * 2) + ((int)(number[x] - '0'));
            bitPos++;
        }
        else
        {
            arr[arrPos] = pomValue;
            arrPos++;
            bitPos = 0;
            pomValue = 0;
        }
    }
    if(bitPos != 0)
    {
        arr[arrPos] = pomValue;
    }
}

__global__ void findPairs(  int *arr, int n, int l)
{
    int id = blockIdx.x * 1024 + threadIdx.x;
    int diff, pom;
    for(int x = id + 1; x < n; x++)
    {
        diff = 0;
        for(int y = 0; y < l; y++)
        {
            pom = 0;
            pom = (arr[(id * l) + y]^arr[(x * l) + y]);    
            while(pom != 0)
            {
                diff += (pom & 1);
                pom = (pom >> 1);
                if(diff > 1)
                    break;
            }
            if(diff > 1)
                break;
        }
        if(diff == 1)
        {
            printf("%d %d\n",id,x);
        }
    }
    
}

int main(int argc, char** argv)
{
    std::ifstream data(argv[1]);
    int n;
    int l;
    data >> n;
    data >> l;
    int bitsPerInt =  15;
    int taken = l / bitsPerInt;
    if(l % bitsPerInt != 0)
        taken++;
    int* arr = new   int[n * taken];
    memset(arr,0,taken * n * sizeof(int));
    std::string number;
    int arrPos = 0;
    while(data >> number)
    {
        parseNumber(&arr[taken * arrPos], number, bitsPerInt);
        arrPos++;
    }
    printf("%d %d\n",n,taken);
    int* arr_d;
    hipMalloc(&arr_d, n * taken * sizeof(int));
    hipMemcpy(arr_d,arr, n * taken * sizeof(int), hipMemcpyHostToDevice);
    int threadCount = 1024;
    int blockSize = (n / threadCount) + 1;
    findPairs<<<blockSize,threadCount>>>(arr_d,n,taken);
    hipDeviceSynchronize();
    data.close();
    hipFree(arr_d);
    delete[] arr;
}