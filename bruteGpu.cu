#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstring>
 
void parseNumber(unsigned int *arr, std::string number)
{
    for(int x = 0; x < number.length();x++)
    {
        arr[x] = (number[x] - '0');
    }
}


__global__ void findPairs(  int *arr, int n, int l)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int diff, pom;
    for(int x = id + 1; x < n; x++)
    {
        diff = 0;
        for(int y = 0; y < l; y++)
        {
           if(arr[x * l + y] != arr[id * l + y])
                diff++;
            if(diff > 1)
                break;
        }
        if(diff == 1)
        {
            printf("%d %d\n",id,x);
        }
    }
    
}

int32_t main(int argc, char** argv)
{
    std::ifstream data(argv[1]);
    int n;
    int l;
    data >> n;
    data >> l;
    unsigned int* arr = new unsigned int[n * l];
    memset(arr,0,n * l * sizeof(unsigned int));
    std::string number;
    int arrPos = 0;

    while(data >> number)
    {
        parseNumber(&arr[arrPos * l], number);
        arrPos++;
    }
    int* arr_d;
    hipMalloc(&arr_d, n * l * sizeof(unsigned int));
    hipMemcpy(arr_d,arr, n * l * sizeof(unsigned int), hipMemcpyHostToDevice);
    int threadCount = 1024;
    int blockSize = n / threadCount + 1;
    findPairs<<<blocks,threadCount>>>(arr_d,n,l);
    data.close();
    hipFree(arr_d);
    delete[] arr;
}