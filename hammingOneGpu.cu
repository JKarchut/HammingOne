#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstring>
#include <sys/time.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void parseNumber(int *arr, std::string number, int bitsPerInt)
{
    int bitPos = 0;
    int arrPos = 0;
    int pomValue = 0;
    for(int x = 0; x < number.length();x++)
    {
        if(bitsPerInt > bitPos)
        {
            pomValue += (int)(number[x] - '0') << bitPos;
            bitPos++;
        }
        else
        {
            arr[arrPos] = pomValue;
            arrPos++;
            bitPos = 0;
            pomValue = 0;
            x--;
        }
    }
    if(bitPos != 0)
    {
        arr[arrPos] = pomValue;
    }
}

__global__ void findPairs(int *arr, int n, int l)
{
    long index = blockIdx.x * blockDim.x + threadIdx.x;
    int diff, pom;
    for(long x = index + 1; x < n; x++)
    {
        diff = 0;
        for(int y = 0; y < l; y++)
        {
            pom = (arr[index * l + y]^arr[x * l + y]);
            if(pom != 0 && (pom & (pom - 1)) == 0)
            {
                diff++;
            }
            else if(pom > 0)
            {
                diff = 2;
            }
            if(diff > 1)
            {
                break;
            }
        }
        if(diff == 1)
        {
            printf("%ld %ld\n",index,x);
        }
    }
}

double GetElapsed(struct timeval begin, struct timeval end)
{
    long seconds = end.tv_sec - begin.tv_sec;
    long microseconds = end.tv_usec - begin.tv_usec;
    return (seconds + microseconds*1e-6) * 1000;
}

int32_t main(int argc, char** argv)
{
    std::ifstream data(argv[1]);
    std::ofstream measures(argv[2], std::ios::app);
    if(arc < 3) return -1;
    struct timeval begin, end;
    int n;
    int l;
    data >> n;
    data >> l;
    int bitsPerInt =  (sizeof(int) * 8) - 1;
    int taken = l / bitsPerInt;
    if(l % bitsPerInt != 0)
        taken++;
    int *arr;
    arr = new int[taken * n];
    memset(arr,0,(long)taken * n * sizeof(int));
    std::string number;
    int arrPos = 0;
    gettimeofday(&begin, 0);
    while(data >> number)
    {
        parseNumber(&arr[(long)arrPos * taken], number, bitsPerInt);
        arrPos++;
    }
    gettimeofday(&end, 0);
    
    measures <<"GPU read data: " << GetElapsed(begin,end) << "ms " << std::endl;
    
    int threadCount = 1024;
    int blockCount = (n / 1024) + 1;
    int *arr_d;
    gettimeofday(&begin, 0);
    gpuErrchk(hipMalloc(&arr_d, (long)taken * n * sizeof(int)));
    gpuErrchk(hipMemcpy(arr_d, arr, (long)taken * n * sizeof(int), hipMemcpyHostToDevice));
    gettimeofday(&end, 0);
    measures <<"GPU alloc and copy to device: " << GetElapsed(begin,end) << "ms " << std::endl;
    
    gpuErrchk(hipDeviceSetLimit(hipLimitPrintfFifoSize, (long long)1e15));
    gettimeofday(&begin, 0);
    findPairs<<<blockCount,threadCount>>>(arr_d,n,taken);
    gpuErrchk( hipPeekAtLastError());
    gpuErrchk( hipDeviceSynchronize());
    gettimeofday(&end, 0);
    measures <<"GPU algorithm: " << GetElapsed(begin,end) << "ms " << std::endl;
    data.close();
    measures.close();
    hipFree(arr_d);
    delete[] arr;
    return 0;
}